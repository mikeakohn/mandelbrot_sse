
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>

#define WIDTH 1024
#define HEIGHT 768
#define CORE_COUNT 128

static int colors[] =
{
  0xff0000,  // f
  0xee3300,  // e
  0xcc5500,  // d
  0xaa5500,  // c
  0xaa3300,  // b
  0x666600,  // a
  0x999900,  // 9
  0x669900,  // 8
  0x339900,  // 7
  0x0099aa,  // 6
  0x0066aa,  // 5
  0x0033aa,  // 4
  0x0000aa,  // 3
  0x000099,  // 2
  0x000066,  // 1
  0x000000,  // 0
};

struct MandelInfo
{
  float r_step;
  float i_step;
  float real_start;
  float imaginary_start;
  int width;
  int height;
};

struct CoreInfo
{
  MandelInfo mandel_info;
  int *picture;
  uint8_t signal_start;
  uint8_t signal_done;
  int core_id;
};

int mandel_calc(
  int *picture,
  int width,
  int height,
  float real_start,
  float real_end,
  float imaginary_start,
  float imaginary_end)
{
  const int max_count = 127;
  int x, y;
  float r, i, r_step, i_step;
  float tr, ti, zr, zi;
  int ptr, count;

  r_step = (real_end - real_start) / (float)width;
  i_step = (imaginary_end - imaginary_start) / (float)height;
  ptr = 0;

  //printf("step = %f %f\n", r_step, i_step);

  i = imaginary_start;

  for (y = 0; y < height; y++)
  {
    r = real_start;

    for (x = 0; x < width; x++)
    {
      zr = 0;
      zi = 0;

      for (count = 0; count < max_count; count++)
      {
        tr = ((zr * zr) - (zi * zi));
        ti = (2 * zr * zi);
        zr = tr + r;
        zi = ti + i;
        if ((zr * zr) + (zi * zi) > 4) break;
      }

      picture[ptr] = colors[count >> 3];

      ptr++;
      r = r + r_step;
    }

    i = i + i_step;
  }

  return 0;
}

__global__
void mandel_calc_cuda_single(
  int *picture,
  int width,
  int height,
  float real_start,
  float real_end,
  float imaginary_start,
  float imaginary_end)
{
  const int max_count = 127;
  int x,y;
  float r, i, r_step, i_step;
  float tr, ti, zr, zi;
  int ptr, count;

  int colors[] =
  {
    0xff0000,  // f
    0xee3300,  // e
    0xcc5500,  // d
    0xaa5500,  // c
    0xaa3300,  // b
    0x666600,  // a
    0x999900,  // 9
    0x669900,  // 8
    0x339900,  // 7
    0x0099aa,  // 6
    0x0066aa,  // 5
    0x0033aa,  // 4
    0x0000aa,  // 3
    0x000099,  // 2
    0x000066,  // 1
    0x000000,  // 0
  };

  r_step = (real_end - real_start) / (float)width;
  i_step = (imaginary_end - imaginary_start) / (float)height;
  ptr = 0;

//printf("step = %f %f\n", r_step, i_step);

  i = imaginary_start;

  for (y = 0; y < height; y++)
  {
    r = real_start;

    for (x = 0; x < width; x++)
    {
      zr = 0;
      zi = 0;

      for (count = 0; count < max_count; count++)
      {
        tr = ((zr * zr) - (zi * zi));
        ti = (2 * zr * zi);
        zr = tr + r;
        zi = ti + i;
        if ((zr * zr) + (zi * zi) > 4) break;
      }

      picture[ptr] = colors[count >> 3];

      ptr++;
      r = r + r_step;
    }

    i = i + i_step;
  }
}

__global__
void mandel_calc_cuda_multi(
  int *picture,
  int width,
  int height,
  float real_start,
  float real_end,
  float imaginary_start,
  float imaginary_end)
{
  const int max_count = 127;
  int x,y;
  float r, i, r_step, i_step;
  float tr, ti, zr, zi;
  int ptr, count;

  int index = threadIdx.x;
  //int stride = blockDim.x;

//printf("index=%d stride=%d\n", index, stride);

  int colors[] =
  {
    0xff0000,  // f
    0xee3300,  // e
    0xcc5500,  // d
    0xaa5500,  // c
    0xaa3300,  // b
    0x666600,  // a
    0x999900,  // 9
    0x669900,  // 8
    0x339900,  // 7
    0x0099aa,  // 6
    0x0066aa,  // 5
    0x0033aa,  // 4
    0x0000aa,  // 3
    0x000099,  // 2
    0x000066,  // 1
    0x000000,  // 0
  };

  r_step = (real_end - real_start) / (float)width;
  i_step = (imaginary_end - imaginary_start) / (float)height;

//printf("step = %f %f\n", r_step, i_step);

  height = height / CORE_COUNT;

  y = index * height;

  ptr = y * width;

  i = imaginary_start + (i_step * y);
#if 0
  i = imaginary_start;

  for (y = 0; y < index * height; y++)
  {
    i = i + i_step;
  }
#endif

//printf("ptr=%d index=%d height=%d %f\n", ptr, index, height, i);

  for (y = 0; y < height; y++)
  {
    r = real_start;

    for (x = 0; x < width; x++)
    {
      zr = 0;
      zi = 0;

      for (count = 0; count < max_count; count++)
      {
        tr = ((zr * zr) - (zi * zi));
        ti = (2 * zr * zi);
        zr = tr + r;
        zi = ti + i;
        if ((zr * zr) + (zi * zi) > 4) break;
      }

      picture[ptr] = colors[count >> 3];

      ptr++;
      r = r + r_step;
    }

    i = i + i_step;
  }
}

int write_int32(FILE *out, int n)
{
  putc((n & 0xff), out);
  putc(((n >> 8) & 0xff), out);
  putc(((n >> 16) & 0xff), out);
  putc(((n >> 24) & 0xff), out);

  return 0;
}

int write_int16(FILE *out, int n)
{
  putc((n & 0xff), out);
  putc(((n >> 8) & 0xff), out);

  return 0;
}

void write_bmp(int *picture, int width, int height)
{
  FILE *out;
  int bmp_width;
  int bmp_size;
  int padding;
  int offset;
  int color;
  int x,y;

  out = fopen("out.bmp", "wb");
  if (out == NULL)
  {
    printf("Can't open file for writing.");
    return;
  }

  bmp_width = width * 3;
  bmp_width = (bmp_width + 3) & (~0x3);
  bmp_size = (bmp_width * height) + 14 + 40;
  padding = bmp_width - (width * 3);

  //printf("width=%d (%d)\n", width, width*3);
  //printf("bmp_width=%d\n", bmp_width);
  //printf("bmp_size=%d\n", bmp_size);

  /* size: 14 bytes */

  putc('B', out);
  putc('M', out);
  write_int32(out, bmp_size);
  write_int16(out, 0);
  write_int16(out, 0);
  write_int32(out, 54);

  /* head1: 14  head2: 40 */

  write_int32(out, 40);         /* biSize */
  write_int32(out, width);
  write_int32(out, height);
  write_int16(out, 1);
  write_int16(out, 24);
  write_int32(out, 0);          /* compression */
  write_int32(out, bmp_width*height);
  write_int32(out, 0);          /* biXPelsperMetre */
  write_int32(out, 0);          /* biYPelsperMetre */
  write_int32(out, 0);
  write_int32(out, 0);

  for (y = 0; y < height; y++)
  {
    offset = y * width;

    for (x = 0; x < width; x++)
    {
      color = picture[offset++];

      putc(color & 0xff, out);
      putc((color >> 8) & 0xff, out);
      putc((color >> 16) & 0xff, out);
    }
    for (x = 0; x < padding; x++) { putc(0, out); }
  }

  fclose(out);
}

int main(int argc, char *argv[])
{
  struct timeval tv_start, tv_end;
  int *picture;

  float real_start = 0.37 - 0.00;
  float real_end = 0.37 + 0.04;
  float imaginary_start = -0.2166 - 0.02;
  float imaginary_end = -0.2166 + 0.02;

  int do_cuda = 0;

  if (argc != 2)
  {
    printf("Usage: %s <normal/cuda/cuda128>\n", argv[0]);
    exit(0);
  }

  if (strcmp(argv[1], "normal") == 0)
  {
    do_cuda = 0;
  }
    else
  if (strcmp(argv[1], "cuda") == 0)
  {
    do_cuda = 1;
  }
    else
  if (strcmp(argv[1], "cuda128") == 0)
  {
    do_cuda = 2;
  }

  const int length = WIDTH * HEIGHT * sizeof(int);

  switch (do_cuda)
  {
    case 0:
      picture = (int *)malloc(length);
      break;
    case 1:
    case 2:
      hipMallocManaged(&picture, length);
      break;
  }

  gettimeofday(&tv_start, NULL);

  if (do_cuda == 1)
  {
    mandel_calc_cuda_single<<<1,1>>>(picture, WIDTH, HEIGHT, real_start, real_end, imaginary_start, imaginary_end);

    hipDeviceSynchronize();
  }
    else
  if (do_cuda == 2)
  {
    mandel_calc_cuda_multi<<<1,128>>>(picture, WIDTH, HEIGHT, real_start, real_end, imaginary_start, imaginary_end);

    hipDeviceSynchronize();
  }
    else
  {
    mandel_calc(picture, WIDTH, HEIGHT, real_start, real_end, imaginary_start, imaginary_end);
  }

  gettimeofday(&tv_end, NULL);

  printf("%ld %ld\n", tv_end.tv_sec, tv_end.tv_usec);
  printf("%ld %ld\n", tv_start.tv_sec, tv_start.tv_usec);
  long time_diff = tv_end.tv_usec - tv_start.tv_usec;
  while(time_diff < 0) { tv_end.tv_sec--; time_diff += 1000000; }
  time_diff += (tv_end.tv_sec - tv_start.tv_sec) * 1000000;
  printf("time=%f\n", (float)time_diff / 1000000);

  switch (do_cuda)
  {
    case 0:
    {
      write_bmp(picture, WIDTH, HEIGHT);
      free(picture);
      break;
    }
    case 1:
    case 2:
    {
      int *image = (int *)malloc(length);
      hipMemcpy(image, picture, length, hipMemcpyDeviceToHost);
      write_bmp(image, WIDTH, HEIGHT);
      hipFree(picture);
      break;
    }
  }

  return 0;
}

